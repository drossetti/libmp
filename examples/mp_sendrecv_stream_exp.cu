#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <unistd.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mp.h>
#include "test_utils.h"

#define MIN_SIZE 1
#define MAX_SIZE 64*1024
#define ITER_COUNT_SMALL 20
#define ITER_COUNT_LARGE 1
#define WINDOW_SIZE 64 

int comm_size, my_rank, peer;

__global__ void dummy_update_kernel(
                    uint32_t * ptr_to_size, int buf_size,
                    uint32_t * ptr_to_lkey, uint32_t lkey,
                    uintptr_t * ptr_to_addr, void * buf_addr
)
{
        if (0 == threadIdx.x && ptr_to_size != NULL && buf_size != 0) { 
            ptr_to_size[0] = buf_size/2;
        }

        if (1 == threadIdx.x && ptr_to_lkey != NULL) { 
            ptr_to_lkey[0] = lkey;
        }

        if (2 == threadIdx.x && ptr_to_addr != NULL) { 
            ptr_to_addr[0] = buf_addr;
        }

        __syncthreads();
        __threadfence_system();
}

int sr_exchange (MPI_Comm comm, int size, int iter_count, int validate)
{
    int j;
    size_t buf_size, buf_size_exp 
    hipStream_t stream;

    /*application and pack buffers*/
    void *buf = NULL, *sbuf_d = NULL, *rbuf_d = NULL;
    void *sbufexp_d = NULL;
    
    /*mp specific objects*/
    mp_request_t *sreq = NULL;
    mp_request_t *rreq = NULL;
    mp_reg_t sreg, rreg, sreg_exp; 

    buf_size = size*iter_count;
	buf_size_exp = (buf_size/2);
    /*allocating requests*/
    sreq = (mp_request_t *) malloc(iter_count*sizeof(mp_request_t));
    rreq = (mp_request_t *) malloc(iter_count*sizeof(mp_request_t));

    hipHostMalloc(&buf, buf_size);
    memset(buf, 0, buf_size); 

    CUDA_CHECK(hipMalloc((void **)&sbuf_d, buf_size));
    CUDA_CHECK(hipMemset(sbuf_d, 0, buf_size)); 

    CUDA_CHECK(hipMalloc((void **)&sbufexp_d, buf_size));
    CUDA_CHECK(hipMemset(sbufexp_d, 0, buf_size)); 

    CUDA_CHECK(hipMalloc((void **)&rbuf_d, buf_size));
    CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size)); 
 
    CUDA_CHECK(hipStreamCreate(&stream));	

    MP_CHECK(mp_register(sbuf_d, buf_size, &sreg));
    MP_CHECK(mp_register(sbufexp_d, buf_size, &sreg_exp));
    MP_CHECK(mp_register(rbuf_d, buf_size, &rreg));

    struct mp_send_info mp_sinfo;
    MP_CHECK(mp_alloc_send_info(&mp_sinfo, MP_HOSTMEM));
    
    if (validate) {
        CUDA_CHECK(hipMemset(sbuf_d, (my_rank + 1), buf_size));
        CUDA_CHECK(hipMemset(sbufexp_d, (my_rank + 2), buf_size));
        CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size));
    }

    for (j = 0; j < iter_count; j++) {
        if (!my_rank) {
            //First method: update parameters on stream
        	dummy_update_kernel<<<1,3,0,stream>>>(
                                            mp_sinfo.ptr_to_size, 
                                            buf_size,
                                            mp_sinfo.ptr_to_lkey, 
                                            sreg_exp.mr->lkey,
                                            mp_sinfo.ptr_to_addr,
                                            sbufexp_d
                                            );
            CUDA_CHECK(hipGetLastError());
            //Prepare and asynchronousl trigger the send
        	MP_CHECK(mp_isend_on_stream_exp((void *)((uintptr_t)sbuf_d + size*j), size, peer, 
                                            &sreg, &sreq[j], &mp_sinfo, stream));
            //MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[j], stream));
            MP_CHECK(mp_wait_on_stream(&sreq[j], stream));

            MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*j), size, peer, &rreg, &rreq[j]));
            MP_CHECK(mp_wait_on_stream(&rreq[j], stream));
        } else {
            MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*j), size, peer, &rreg, &rreq[j]));
            MP_CHECK(mp_wait_on_stream(&rreq[j], stream));

            //Second method: prepare descriptors
            MP_CHECK(mp_prepare_send_exp((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[j], &mp_sinfo));

            //update parameters on stream
            dummy_update_kernel<<<1,3,0,stream>>>(
                                            mp_sinfo.ptr_to_size, 
                                            buf_size,
                                            mp_sinfo.ptr_to_lkey, 
                                            sreg_exp.mr->lkey,
                                            mp_sinfo.ptr_to_addr,
                                            sbufexp_d
                                            );

            //Trigger the send
            MP_CHECK(mp_post_send_on_stream_exp(peer, &sreq[j], stream));

            //MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[j], stream));
            MP_CHECK(mp_wait_on_stream(&sreq[j], stream));
        }
    } 
    MP_CHECK(mp_wait_all(iter_count, rreq));
    MP_CHECK(mp_wait_all(iter_count, sreq));
    // all ops in the stream should have been completed 
    usleep(1000);
    CUDA_CHECK(hipStreamQuery(stream));
    MPI_CHECK(MPI_Barrier(comm));

    if (validate && my_rank) {
        CUDA_CHECK(hipMemcpy(buf, rbuf_d, buf_size, hipMemcpyDefault));
        char *value = (char*)buf;
        char expected = (char) (peer + 1);
        for (j=0; j<(iter_count*size); j++) {
             if (value[j] != (peer + 1)) {
                fprintf(stderr, "validation check failed index: %d expected: %d actual: %d \n", j, expected, value[j]);
                 exit(-1);
             }
        }
    }
    MPI_CHECK(MPI_Barrier(comm));
    CUDA_CHECK(hipDeviceSynchronize());
    mp_deregister(&sreg);
    mp_deregister(&rreg);
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(sbuf_d));
    CUDA_CHECK(hipFree(rbuf_d));
    hipHostFree(buf);
    free(sreq);
    free(rreq);

    return 0;
}

int main (int c, char *v[])
{
    int iter_count, size;
    int validate = 1;

    MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &comm_size));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &my_rank));

    if (comm_size != 2) { 
	fprintf(stderr, "this test requires exactly two processes \n");
        exit(-1);
    }

    if (gpu_init(-1)) {
        fprintf(stderr, "got error while initializing GPU\n");
        MPI_Abort(MPI_COMM_WORLD, -1);
    }

    peer = !my_rank;
    //Need to set CUDA_VISIBLE_DEVICES
    MP_CHECK(mp_init(MPI_COMM_WORLD, &peer, 1, MP_INIT_DEFAULT, 0));

    iter_count = ITER_COUNT_SMALL;

    for (size=MIN_SIZE; size<=MAX_SIZE; size*=2) 
    {
        if (size > 1024) {
            iter_count = ITER_COUNT_LARGE;
        }

        sr_exchange(MPI_COMM_WORLD, size, iter_count, validate);

        if (!my_rank) fprintf(stdout, "# SendRecv test passed validation with message size: %d \n", size);
    }

    mp_finalize();
    MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    MPI_CHECK(MPI_Finalize());
    return 0;
}
